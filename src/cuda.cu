#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include "constants.hpp"


#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <hiprand/hiprand_kernel.h>

// Kernel function to add the elements of two arrays
__global__
void cuda_fitness(int *data, int *results) {
    int individual = threadIdx.x + blockIdx.x * (THREADS_PER_BLOCK / SIDE);
    int individual_start = individual * SIDE;
    for (int i = 0; i < SIDE; i++) {
        for (int j = i + 1; j < SIDE; j++) {
            if (data[individual_start + i] == data[individual_start + j] ||
                i - data[individual_start + i] == j - data[individual_start + j] ||
                i + data[individual_start + i] == j + data[individual_start + j]) {

                results[individual]++;
            }
        }
    }
}

__global__
void cuda_reproduce(int *data, int* data_next, int *results, int *results_idx, hiprandState *states) {
    int individual = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int individual_start = individual * SIDE;

    int parent1 = (int) (hiprand_uniform(&states[individual]) * TOURNAMENT_BEST);
    int parent2 = (int) (hiprand_uniform(&states[individual]) * TOURNAMENT_BEST);

    int parent1_idx = results[parent1];
    int parent2_idx = results[parent2];

    int parent1_start = parent1_idx * SIDE;
    int parent2_start = parent2_idx * SIDE;


    for (int i = 0; i < SIDE; i++) {
        if (hiprand_uniform(&states[individual]) < MUTATION_PROBABILITY) {
            data_next[individual_start + i] = (int) (hiprand_uniform(&states[individual]) * SIDE);
        } else {
            if (hiprand_uniform(&states[individual]) > 0.5) {
                data_next[individual_start + i] = data[parent1_start + i];
            } else {
                data_next[individual_start + i] = data[parent2_start + i];
            }
        }
    }

    // TODO
}

__global__
void setup_kernel(hiprandState *states)
{
    int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    hiprand_init(id, 0, 0, &states[id]);
}


struct RandGen {
    unsigned int clock;

    RandGen(unsigned int _clock) : clock(_clock) {}

    __device__
    float operator()(int idx) {
        thrust::default_random_engine rng(clock);
        thrust::uniform_int_distribution<int> dist(0, SIDE);
        rng.discard(idx);
        return dist(rng);
    }
};

void run() {
    // Initialize data
    thrust::device_vector<int> data(POPULATION_SIZE * SIDE);
    thrust::device_vector<int> data_next(POPULATION_SIZE * SIDE);
    thrust::device_vector<int> results(POPULATION_SIZE);
    thrust::device_vector<int> results_idx(POPULATION_SIZE);

    // Setups states for hiprand generation
    thrust::device_vector<hiprandState> curand_states(POPULATION_SIZE);
    setup_kernel<<<POPULATION_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(&curand_states[0])
    );

    thrust::transform(
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(POPULATION_SIZE * SIDE),
        data.begin(),
        RandGen(static_cast<unsigned int>(clock()))
    );

    bool finished = false;
    int solution[SIDE];

    int generations = 0;
    int min_fitness;

    // while (!finished) {
    cuda_fitness <<<POPULATION_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (
            thrust::raw_pointer_cast(&data[0]),
            thrust::raw_pointer_cast(&results[0])
    );

    thrust::sequence(results_idx.begin(), results_idx.end());
    thrust::sort_by_key(results.begin(), results.end(), results_idx.begin());

    cuda_reproduce <<<POPULATION_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (
            thrust::raw_pointer_cast(&data[0]),
            thrust::raw_pointer_cast(&data_next[0]),
            thrust::raw_pointer_cast(&results[0]),
            thrust::raw_pointer_cast(&results_idx[0]),
            thrust::raw_pointer_cast(&curand_states[0])
    );

    data.swap(data_next);

    //}

}
