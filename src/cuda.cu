#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include "constants.hpp"

// Kernel function to add the elements of two arrays
__global__
void cuda_fitness(int side, int population, int *data, int *results) {
    int individual = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
    int individual_start = individual * (side + 1);
    for (int i = 0; i < side; i++) {
        for (int j = i; j < side; j++) {
            if (i != j) {
                if (data[individual_start + i] == data[individual_start + j]) {

                } else if() {
                    
                }
            }
        }
    }
}

int *data_gpu;
int *results_gpu;
bool inited = false;

void fitness(int side, int population, int *data, int* results) {

    if (!inited) {
        // Allocate Unified Memory – accessible from CPU or GPU
        hipMalloc(&data_gpu, (side) * population * sizeof(int));
        hipMalloc(&results_gpu, population * sizeof(int));
        inited = true;
    }

    // Copy input data to array on GPU.
    hipMemcpy(data, data_gpu, (side + 1) * population * sizeof(int), hipMemcpyHostToDevice);

    printf("%d", POPULATION_SIZE / THREADS_PER_BLOCK);
    // Run kernel on 1M elements on the GPU
    cuda_fitness <<<POPULATION_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (side, population, data_gpu, results_gpu);

    // Wait for GPU to clearCuda before accessing on host
    hipDeviceSynchronize();

    // Copy results
    hipMemcpy(results_gpu, results, population * sizeof(int), hipMemcpyDeviceToHost);
}

void clearCuda() {
    // Free memory
    hipFree(data_gpu);
    hipFree(results_gpu);
}